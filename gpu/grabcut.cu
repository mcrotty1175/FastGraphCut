#include "hip/hip_runtime.h"
#include <cstring>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <cmath>
#include <iterator>
#include "grabcut.h"
#include "graph.hpp"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <omp.h>
#include <hiprand/hiprand_kernel.h>

#define COMPONENT_COUNT 5
using namespace std;

typedef struct
{
    double *model;

    double *coefs;
    double *mean;
    double *cov;

    double inverseCovs[COMPONENT_COUNT][3][3];
    double covDeterms[COMPONENT_COUNT];

    double sums[COMPONENT_COUNT][3];
    double prods[COMPONENT_COUNT][3][3];
    int sampleCounts[COMPONENT_COUNT];
    int totalSampleCount;
} GMM_t;

void initLearning(GMM_t *gmm);
void addSample(GMM_t *gmm, int ci, pixel_t color);
void endLearning(GMM_t *gmm);
int whichComponent(GMM_t *gmm, pixel_t color);

void calcInverseCovAndDeterm(GMM_t *gmm, int ci, double singularFix);

void initEmptyGMM(GMM_t *gmm)
{
    int modelSize = 3 /*mean*/ + 9 /*covariance*/ + 1 /*component weight*/;
    if (gmm == NULL)
        return;

    // gmm = (GMM_t *)malloc(sizeof(GMM_t));
    gmm->model = (double *)calloc(modelSize * COMPONENT_COUNT, sizeof(double));
    if (gmm->model == NULL)
    {
        std::cerr << "Memory allocation failed!" << std::endl;
        return; // DO WE NEED THIS?
    }

    gmm->coefs = gmm->model;
    gmm->mean = gmm->coefs + COMPONENT_COUNT;
    gmm->cov = gmm->mean + 3 * COMPONENT_COUNT;

    // Pretty sure this doesn't do anything for a new array
    for (int ci = 0; ci < COMPONENT_COUNT; ci++)
        calcInverseCovAndDeterm(gmm, ci, 0.0);

    gmm->totalSampleCount = 0;
}

double getComponent(GMM_t *gmm, int ci, pixel_t color)
{
    double result = 0;
    if (gmm->coefs[ci] > 0)
    {
        double *m = gmm->mean + 3 * ci;
        double diff[3] = {color.r - m[0], color.g - m[1], color.b - m[2]};
        double mult = diff[0] * (diff[0] * gmm->inverseCovs[ci][0][0] + diff[1] * gmm->inverseCovs[ci][1][0] + diff[2] * gmm->inverseCovs[ci][2][0]) + diff[1] * (diff[0] * gmm->inverseCovs[ci][0][1] + diff[1] * gmm->inverseCovs[ci][1][1] + diff[2] * gmm->inverseCovs[ci][2][1]) + diff[2] * (diff[0] * gmm->inverseCovs[ci][0][2] + diff[1] * gmm->inverseCovs[ci][1][2] + diff[2] * gmm->inverseCovs[ci][2][2]);
        result = 1.0f / sqrt(gmm->covDeterms[ci]) * exp(-0.5f * mult);
    }
    return result;
}

double doSomething(GMM_t *gmm, pixel_t color)
{
    double res = 0;
    for (int ci = 0; ci < COMPONENT_COUNT; ci++)
        res += gmm->coefs[ci] * getComponent(gmm, ci, color);
    return res;
}

int whichComponent(GMM_t *gmm, pixel_t color)
{
    int k = 0;
    double max = 0;

    for (int ci = 0; ci < COMPONENT_COUNT; ci++)
    {
        double p = getComponent(gmm, ci, color);
        if (p > max)
        {
            k = ci;
            max = p;
        }
    }

    return k;
}

void initLearning(GMM_t *gmm)
{
    if (gmm == NULL)
        return;

    for (int ci = 0; ci < COMPONENT_COUNT; ci++)
    {
        gmm->sums[ci][0] = gmm->sums[ci][1] = gmm->sums[ci][2] = 0;
        gmm->prods[ci][0][0] = gmm->prods[ci][0][1] = gmm->prods[ci][0][2] = 0;
        gmm->prods[ci][1][0] = gmm->prods[ci][1][1] = gmm->prods[ci][1][2] = 0;
        gmm->prods[ci][2][0] = gmm->prods[ci][2][1] = gmm->prods[ci][2][2] = 0;
        gmm->sampleCounts[ci] = 0;
    }
    gmm->totalSampleCount = 0;
}

void addSample(GMM_t *gmm, int ci, pixel_t color)
{
    if (gmm == NULL)
    {
        cout << "gmm is null in addsample\n";
        return;
    }

    if (ci < 0 || ci >= COMPONENT_COUNT)
    {
        std::cerr << "Invalid component index in addSample: " << ci << std::endl;
        return;
    }

    gmm->sums[ci][0] += color.r;
    gmm->sums[ci][1] += color.g;
    gmm->sums[ci][2] += color.b;
    gmm->prods[ci][0][0] += color.r * color.r;
    gmm->prods[ci][0][1] += color.r * color.g;
    gmm->prods[ci][0][2] += color.r * color.b;
    gmm->prods[ci][1][0] += color.g * color.r;
    gmm->prods[ci][1][1] += color.g * color.g;
    gmm->prods[ci][1][2] += color.g * color.b;
    gmm->prods[ci][2][0] += color.b * color.r;
    gmm->prods[ci][2][1] += color.b * color.g;
    gmm->prods[ci][2][2] += color.b * color.b;
    gmm->sampleCounts[ci]++;
    gmm->totalSampleCount++;
}

void endLearning(GMM_t *gmm)
{
    if (gmm == NULL)
        return;
    for (int ci = 0; ci < COMPONENT_COUNT; ci++)
    {
        int n = gmm->sampleCounts[ci];
        if (n == 0)
            gmm->coefs[ci] = 0;
        else
        {
            double inv_n = 1.0 / n;
            gmm->coefs[ci] = (double)n / gmm->totalSampleCount;

            double *m = gmm->mean + 3 * ci;
            m[0] = gmm->sums[ci][0] * inv_n;
            m[1] = gmm->sums[ci][1] * inv_n;
            m[2] = gmm->sums[ci][2] * inv_n;

            double *c = gmm->cov + 9 * ci;
            c[0] = gmm->prods[ci][0][0] * inv_n - m[0] * m[0];
            c[1] = gmm->prods[ci][0][1] * inv_n - m[0] * m[1];
            c[2] = gmm->prods[ci][0][2] * inv_n - m[0] * m[2];
            c[3] = gmm->prods[ci][1][0] * inv_n - m[1] * m[0];
            c[4] = gmm->prods[ci][1][1] * inv_n - m[1] * m[1];
            c[5] = gmm->prods[ci][1][2] * inv_n - m[1] * m[2];
            c[6] = gmm->prods[ci][2][0] * inv_n - m[2] * m[0];
            c[7] = gmm->prods[ci][2][1] * inv_n - m[2] * m[1];
            c[8] = gmm->prods[ci][2][2] * inv_n - m[2] * m[2];

            calcInverseCovAndDeterm(gmm, ci, 0.01);
        }
    }
    // Print GMM means
    // std::cout << "GMM Means:" << std::endl;
    // for (int ci = 0; ci < COMPONENT_COUNT; ci++)
    // {
    //     double *m = gmm->mean + 3 * ci;
    //     std::cout << "Component " << ci << ": (" << m[0] << ", " << m[1] << ", " << m[2] << ")" << std::endl;
    // }

    // Print GMM covariance matrices
    // std::cout << "GMM Covariance Matrices:" << std::endl;
    // for (int ci = 0; ci < COMPONENT_COUNT; ci++)
    // {
    //     double *c = gmm->cov + 9 * ci;
    //     std::cout << "Component " << ci << ":" << std::endl;
    //     std::cout << "[" << c[0] << ", " << c[1] << ", " << c[2] << "]" << std::endl;
    //     std::cout << "[" << c[3] << ", " << c[4] << ", " << c[5] << "]" << std::endl;
    //     std::cout << "[" << c[6] << ", " << c[7] << ", " << c[8] << "]" << std::endl;
    // }
}

void calcInverseCovAndDeterm(GMM_t *gmm, int ci, double singularFix)
{
    if (gmm == NULL)
        return;

    if (gmm->coefs[ci] > 0)
    {
        double *c = gmm->cov + 9 * ci;
        double dtrm = c[0] * (c[4] * c[8] - c[5] * c[7]) - c[1] * (c[3] * c[8] - c[5] * c[6]) + c[2] * (c[3] * c[7] - c[4] * c[6]);
        if (dtrm <= 1e-6 && singularFix > 0)
        {
            // Adds the white noise to avoid singular covariance matrix.
            c[0] += singularFix;
            c[4] += singularFix;
            c[8] += singularFix;
            dtrm = c[0] * (c[4] * c[8] - c[5] * c[7]) - c[1] * (c[3] * c[8] - c[5] * c[6]) + c[2] * (c[3] * c[7] - c[4] * c[6]);
        }
        gmm->covDeterms[ci] = dtrm;

        double inv_dtrm = 1.0 / dtrm;
        gmm->inverseCovs[ci][0][0] = (c[4] * c[8] - c[5] * c[7]) * inv_dtrm;
        gmm->inverseCovs[ci][1][0] = -(c[3] * c[8] - c[5] * c[6]) * inv_dtrm;
        gmm->inverseCovs[ci][2][0] = (c[3] * c[7] - c[4] * c[6]) * inv_dtrm;
        gmm->inverseCovs[ci][0][1] = -(c[1] * c[8] - c[2] * c[7]) * inv_dtrm;
        gmm->inverseCovs[ci][1][1] = (c[0] * c[8] - c[2] * c[6]) * inv_dtrm;
        gmm->inverseCovs[ci][2][1] = -(c[0] * c[7] - c[1] * c[6]) * inv_dtrm;
        gmm->inverseCovs[ci][0][2] = (c[1] * c[5] - c[2] * c[4]) * inv_dtrm;
        gmm->inverseCovs[ci][1][2] = -(c[0] * c[5] - c[2] * c[3]) * inv_dtrm;
        gmm->inverseCovs[ci][2][2] = (c[0] * c[4] - c[1] * c[3]) * inv_dtrm;
    }
}

static double calcBeta(image_t *img)
{
    double beta = 0;
    for (int y = 0; y < img->rows; y++)
    {
        for (int x = 0; x < img->cols; x++)
        {
            if (x > 0)
                beta += dot_diff_rgb(get_r(img, y, x), get_g(img, y, x), get_b(img, y, x), get_r(img, y, x - 1), get_g(img, y, x - 1), get_b(img, y, x - 1));
            if (y > 0 && x > 0)
                beta += dot_diff_rgb(get_r(img, y, x), get_g(img, y, x), get_b(img, y, x), get_r(img, y - 1, x - 1), get_g(img, y - 1, x - 1), get_b(img, y - 1, x - 1));
            if (y > 0)
                beta += dot_diff_rgb(get_r(img, y, x), get_g(img, y, x), get_b(img, y, x), get_r(img, y - 1, x), get_g(img, y - 1, x), get_b(img, y - 1, x));
            if (y > 0 && x < img->cols - 1)
                beta += dot_diff_rgb(get_r(img, y, x), get_g(img, y, x), get_b(img, y, x), get_r(img, y - 1, x + 1), get_g(img, y - 1, x + 1), get_b(img, y - 1, x + 1));
        }
    }

    if (beta <= 0.0000001f)
        beta = 0;
    else
        beta = 1.f / (2 * beta / (4 * img->cols * img->rows - 3 * img->cols - 3 * img->rows + 2));

    return beta;
}

static void calcNWeights(image_t *img, double *leftW, double *upleftW, double *upW, double *uprightW, double beta, double gamma)
{
    double gammaDivSqrt2 = gamma / sqrt(2.0);
    uint64_t num_pixels = img->rows * img->cols;

    for (int y = 0; y < img->rows; y++)
    {
        for (int x = 0; x < img->cols; x++)
        {
            int idx = y * img->cols + x;
            if (x > 0)
                leftW[idx] = gamma * exp(-beta * dot_diff_rgb(get_r(img, y, x), get_g(img, y, x), get_b(img, y, x), get_r(img, y, x - 1), get_g(img, y, x - 1), get_b(img, y, x - 1)));
            else
                leftW[idx] = 0;
            if (x > 0 && y > 0)
                upleftW[idx] = gammaDivSqrt2 * exp(-beta * dot_diff_rgb(get_r(img, y, x), get_g(img, y, x), get_b(img, y, x), get_r(img, y - 1, x - 1), get_g(img, y - 1, x - 1), get_b(img, y - 1, x - 1)));
            else
                upleftW[idx] = 0;
            if (y > 0)
                upW[idx] = gamma * exp(-beta * dot_diff_rgb(get_r(img, y, x), get_g(img, y, x), get_b(img, y, x), get_r(img, y - 1, x), get_g(img, y - 1, x), get_b(img, y - 1, x)));
            else
                upW[idx] = 0;
            if (x < img->cols - 1 && y > 0)
                uprightW[idx] = gammaDivSqrt2 * exp(-beta * dot_diff_rgb(get_r(img, y, x), get_g(img, y, x), get_b(img, y, x), get_r(img, y - 1, x + 1), get_g(img, y - 1, x + 1), get_b(img, y - 1, x + 1)));
            else
                uprightW[idx] = 0;
        }
    }
}

// Technically should have a checkMask fn

static void initMaskWithRect(mask_t *mask, rect_t rect, image_t *img)
{
    mask->rows = img->rows;
    mask->cols = img->cols;
    mask->array = (MaskVal *)calloc(img->rows * img->cols, sizeof(MaskVal));

    int start_x = rect.x;
    int start_y = rect.y;

    int remaining_width = img->cols - start_x;
    int width = (rect.width < remaining_width) ? rect.width : remaining_width;

    int remaining_height = img->rows - start_y;
    int end_y = rect.height < remaining_height ? rect.height : remaining_height;
    end_y += start_y;
    int margin = 15;
    for (int r = start_y; r < end_y; r++)
    {
        for (int c = start_x; c < start_x + width; c++)
        {
            // if (r > start_y + margin && r < end_y - margin && c > start_x + margin && c < start_x + width - margin)
            //     mask->array[r * img->cols + c] = GC_FGD;
            // else
            //     mask->array[r * img->cols + c] = GC_PR_FGD;

            mask->array[r * img->cols + c] = GC_PR_FGD;
        }
    }
}

/*
void kmeans(pixel_t *pixels, int num_pixels, int k, int num_clusters, int max_iters, int *labels)
{
    // labels = (int *)malloc(num_pixels * sizeof(int));
    // Allocate centroids
    Centroid *centroids = (Centroid *)malloc(num_clusters * sizeof(Centroid));
    Centroid *new_centroids = (Centroid *)malloc(num_clusters * sizeof(Centroid));
    int *counts = (int *)malloc(num_clusters * sizeof(int));

    // Set initial cluster centers randomly
    for (int i = 0; i < num_clusters; ++i)
    {
        int idx = rand() % num_pixels;
        centroids[i].r = pixels[idx].r;
        centroids[i].g = pixels[idx].g;
        centroids[i].b = pixels[idx].b;
    }

    for (int iter = 0; iter < max_iters; ++iter)
    {
        // Reset accumulators
        for (int i = 0; i < num_clusters; ++i)
        {
            new_centroids[i].r = 0;
            new_centroids[i].g = 0;
            new_centroids[i].b = 0;
            counts[i] = 0;
        }

        // Assign labels based on nearest centroid
        for (int i = 0; i < num_pixels; ++i)
        {
            float min_dist = INFINITY;
            int label = 0;
            for (int j = 0; j < num_clusters; ++j)
            {
                float dist = distance_squared(pixels[i], centroids[j]);
                if (dist < min_dist)
                {
                    min_dist = dist;
                    label = j;
                }
            }
            labels[i] = label;
            new_centroids[label].r += pixels[i].r;
            new_centroids[label].g += pixels[i].g;
            new_centroids[label].b += pixels[i].b;
            counts[label]++;
        }

        // Update centroids
        int converged = 1;
        for (int i = 0; i < num_clusters; ++i)
        {
            if (counts[i] == 0)
                continue; // avoid division by zero

            Centroid updated = {
                new_centroids[i].r / counts[i],
                new_centroids[i].g / counts[i],
                new_centroids[i].b / counts[i]};

            // Check if centroid has changed significantly
            // pixel_t estimate_center = {(uint8_t)centroids[i].r, (uint8_t)centroids[i].g, (uint8_t)centroids[i].b};
            // float shift = distance_squared(estimate_center, updated);

            float shift =
                (centroids[i].r - updated.r) * (centroids[i].r - updated.r) +
                (centroids[i].g - updated.g) * (centroids[i].g - updated.g) +
                (centroids[i].b - updated.b) * (centroids[i].b - updated.b);

            if (shift > 1e-4f)
            {
                converged = 0;
            }

            centroids[i] = updated;
        }

        if (converged)
            break;
    }

    free(centroids);
    free(new_centroids);
    free(counts);
}
*/

__global__ void kmeans_gpu(
    uint8_t *r, uint8_t *g, uint8_t *b, int num_pixels,
    Centroid *centroids, Centroid *new_centroids, int *counts,
    int *labels, int num_clusters, int max_iters)
{
    int block_id = blockIdx.x;

    int num_bytes = 256;                     // use for getting parts of global image
    __shared__ Centroid shared_centroids[5]; // = centroids;
    __shared__ float local_sum_r[5];
    __shared__ float local_sum_g[5];
    __shared__ float local_sum_b[5];
    __shared__ int local_count[5];
    // __shared__ uint8_t red[num_bytes] = r[block_id * num_bytes];
    //__shared__ uint8_t green[num_bytes] = g[block_id * num_bytes];
    // __shared__ uint8_t blue[num_bytes] = b[block_id * num_bytes];

    // float* s_c = buffer;
    // float* s_a = buffer + 4096;
    // float* s_b = buffer + 4096 * 2;
    // cout << "number of threads: " << blockDim.x << endl;

    int id = blockIdx.x * blockDim.x + threadIdx.x; // and/or y
    int tid = threadIdx.x;                          // thread id within block

    if (id == 0)
        printf("in gpu %d\n", num_pixels);
    if (id >= num_pixels)
        return;

    for (int iter = 0; iter < max_iters; ++iter)
    {
        if (tid == 0)
        {
            for (int i = 0; i < num_clusters; i++)
            {
                shared_centroids[tid] = centroids[tid];
                local_sum_r[tid] = 0.0f;
                local_sum_g[tid] = 0.0f;
                local_sum_b[tid] = 0.0f;
                local_count[tid] = 0;
            }
        }
        __syncthreads();

        float ri = r[id];
        float gi = g[id];
        float bi = b[id];

        float min_dist = INFINITY;
        int label = 0;

        for (int j = 0; j < num_clusters; ++j)
        {
            float dist = (ri - shared_centroids[j].r) * (ri - shared_centroids[j].r) + (gi - shared_centroids[j].g) * (gi - shared_centroids[j].g) + (bi - shared_centroids[j].b) * (bi - shared_centroids[j].b);
            if (dist < min_dist)
            {
                min_dist = dist;
                label = j;
            }
        }
        labels[id] = label;

        atomicAdd(&local_sum_r[label], ri);
        atomicAdd(&local_sum_g[label], gi);
        atomicAdd(&local_sum_b[label], bi);
        atomicAdd(&local_count[label], 1);
        __syncthreads(); // unncessary?

        if (tid == 0)
        {
            for (int i = 0; i < num_clusters; i++)
            {
                if (local_count[tid] == 0)
                    continue; // avoid division by zero
                centroids[i].r = local_sum_r[tid] / local_count[tid];
                centroids[i].g = local_sum_g[tid] / local_count[tid];
                centroids[i].b = local_sum_b[tid] / local_count[tid];
            }
        }

        __syncthreads();
    }
}

/*
  Initialize GMM background and foreground models using kmeans algorithm.
*/
static void initGMMs(image_t *img, mask_t *mask, GMM_t *bgdGMM, GMM_t *fgdGMM)
{

    // More realistically, we should only definitely put the kmean's num_pixels for loop in the kernel, not entire kmeans algorithm
    int kMeansItCount = 10;
    int k = 5;
    std::vector<uint8_t> bgdR, bgdG, bgdB;
    std::vector<uint8_t> fgdR, fgdG, fgdB;

    for (int r = 0; r < img->rows; r++)
    {
        for (int c = 0; c < img->cols; c++)
        {
            MaskVal m = mask_at(mask, r, c);
            if (m == GC_BGD || m == GC_PR_BGD)
            {
                bgdR.push_back(get_r(img, r, c));
                bgdG.push_back(get_g(img, r, c));
                bgdB.push_back(get_b(img, r, c));
            }

            // GC_FGD | GC_PR_FGD
            else
            {
                fgdR.push_back(get_r(img, r, c));
                fgdG.push_back(get_g(img, r, c));
                fgdB.push_back(get_b(img, r, c));
            }
        }
    }

    int bdg_size = bgdR.size();
    int fgd_size = fgdR.size();

    int *bgdLabels = (int *)malloc(bdg_size * sizeof(int));
    int *fgdLabels = (int *)malloc(fgd_size * sizeof(int));

    {
        int num_clusters = std::min(COMPONENT_COUNT, bdg_size);

        uint8_t *d_bgdR, *d_bgdG, *d_bgdB;
        hipMalloc((void **)&d_bgdR, bdg_size * sizeof(uint8_t));
        hipMalloc((void **)&d_bgdG, bdg_size * sizeof(uint8_t));
        hipMalloc((void **)&d_bgdB, bdg_size * sizeof(uint8_t));

        hipMemcpy(d_bgdR, bgdR.data(), bdg_size * sizeof(uint8_t), hipMemcpyHostToDevice);
        hipMemcpy(d_bgdG, bgdG.data(), bdg_size * sizeof(uint8_t), hipMemcpyHostToDevice);
        hipMemcpy(d_bgdB, bgdB.data(), bdg_size * sizeof(uint8_t), hipMemcpyHostToDevice);

        Centroid *centroids = (Centroid *)malloc(num_clusters * sizeof(Centroid)); // in kmeans
        Centroid *new_centroids;
        int *counts;
        int *dev_bgdLabels;
        Centroid *dev_centroids;

        hipMalloc((void **)&dev_centroids, num_clusters * sizeof(Centroid));
        hipMalloc((void **)&new_centroids, num_clusters * sizeof(Centroid));
        hipMalloc((void **)&counts, num_clusters * sizeof(int));
        hipMalloc((void **)&dev_bgdLabels, bdg_size * sizeof(int));

        srand(1);
        for (int i = 0; i < num_clusters; i++)
        {
            int idx = rand() % (bdg_size);
            centroids[i].r = img->r[idx];
            centroids[i].g = img->g[idx];
            centroids[i].b = img->b[idx];
        }
        hipMemcpy(dev_centroids, centroids, num_clusters * sizeof(Centroid), hipMemcpyHostToDevice);

        cout << "before bgd num pixels: " << bdg_size << endl;
        int threadsPerBlock = 1024;
        int numBlocks = (bdg_size + threadsPerBlock - 1) / (threadsPerBlock);
        auto start = std::chrono::high_resolution_clock::now();
        kmeans_gpu<<<numBlocks, threadsPerBlock>>>(d_bgdR, d_bgdG, d_bgdB, bdg_size,
                                                   dev_centroids, new_centroids, counts, dev_bgdLabels, num_clusters, kMeansItCount);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

        cout << "K-means for background took: " << duration.count() << " us\n";
        cout << "hiii\n";

        hipMemcpy(bgdLabels, dev_bgdLabels, bdg_size * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_bgdR);
        hipFree(d_bgdG);
        hipFree(d_bgdB);
        hipFree(dev_centroids);
        hipFree(new_centroids);
        hipFree(counts);
        hipFree(dev_bgdLabels);
        free(centroids);
    }

    {
        int num_clusters = std::min(COMPONENT_COUNT, fgd_size);

        uint8_t *d_fgdR, *d_fgdG, *d_fgdB;
        hipMalloc((void **)&d_fgdR, fgd_size * sizeof(uint8_t));
        hipMalloc((void **)&d_fgdG, fgd_size * sizeof(uint8_t));
        hipMalloc((void **)&d_fgdB, fgd_size * sizeof(uint8_t));

        hipMemcpy(d_fgdR, fgdR.data(), fgd_size * sizeof(uint8_t), hipMemcpyHostToDevice);
        hipMemcpy(d_fgdG, fgdG.data(), fgd_size * sizeof(uint8_t), hipMemcpyHostToDevice);
        hipMemcpy(d_fgdB, fgdB.data(), fgd_size * sizeof(uint8_t), hipMemcpyHostToDevice);

        Centroid *f_centroids = (Centroid *)malloc(num_clusters * sizeof(Centroid)); // in kmeans
        Centroid *new_centroids;
        int *counts;
        int *dev_fgdLabels;
        Centroid *dev_centroids;

        hipMalloc((void **)&dev_centroids, num_clusters * sizeof(Centroid));
        hipMalloc((void **)&new_centroids, num_clusters * sizeof(Centroid));
        hipMalloc((void **)&counts, num_clusters * sizeof(int));
        hipMalloc((void **)&dev_fgdLabels, fgd_size * sizeof(int));

        srand(1);
        for (int i = 0; i < num_clusters; i++)
        {
            int idx = rand() % (fgd_size);
            f_centroids[i].g = img->g[idx];
            f_centroids[i].r = img->r[idx];
            f_centroids[i].b = img->b[idx];
        }
        hipMemcpy(dev_centroids, f_centroids, num_clusters * sizeof(Centroid), hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int numBlocks = (fgd_size + threadsPerBlock - 1) / (threadsPerBlock);
        std::cout << "before fgd num pixels " << fgd_size << endl;
        ;
        auto start = std::chrono::high_resolution_clock::now();
        kmeans_gpu<<<numBlocks, threadsPerBlock>>>(d_fgdR, d_fgdG, d_fgdB, fgd_size,
                                                   dev_centroids, new_centroids, counts, dev_fgdLabels, num_clusters, kMeansItCount);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

        cout << "K-means for foreground took: " << duration.count() << " us\n";

        hipMemcpy(fgdLabels, dev_fgdLabels, fgd_size * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_fgdR);
        hipFree(d_fgdG);
        hipFree(d_fgdB);
        hipFree(dev_centroids);
        hipFree(new_centroids);
        hipFree(counts);
        hipFree(dev_fgdLabels);
        free(f_centroids);
    }

    cout << "done with kmeans\n";
    initLearning(bgdGMM);
    for (int i = 0; i < bdg_size; i++)
    {
        pixel_t px = {bgdR[i], bgdG[i], bgdB[i]};
        addSample(bgdGMM, bgdLabels[i], px);
    }
    // std::cout << "BGD GMM means weights after initGMMs" << std::endl;
    endLearning(bgdGMM);

    initLearning(fgdGMM);
    for (int i = 0; i < fgd_size; i++)
    {
        pixel_t px = {fgdR[i], fgdG[i], fgdB[i]};
        addSample(fgdGMM, fgdLabels[i], px);
    }
    // std::cout << "FGD GMM means weights after initGMMs" << std::endl;
    endLearning(fgdGMM);
}

static void assignGMMsComponents(image_t *img, mask_t *mask, GMM_t *bgdGMM, GMM_t *fgdGMM, int *compIdxs)
{
    for (int r = 0; r < img->rows; r++)
    {
        int row_index = r * img->cols;
        for (int c = 0; c < img->cols; c++)
        {
            pixel_t color = {get_r(img, r, c), get_g(img, r, c), get_b(img, r, c)};
            MaskVal m = mask_at(mask, r, c);
            compIdxs[row_index + c] = (m == GC_BGD || m == GC_PR_BGD) ? whichComponent(bgdGMM, color) : whichComponent(fgdGMM, color);
        }
    }
}

/*
  Learn GMMs parameters.
*/
static void learnGMMs(image_t *img, mask_t *mask, int *compIdxs, GMM_t *bgdGMM, GMM_t *fgdGMM, int iter)
{
    initLearning(bgdGMM);
    initLearning(fgdGMM);
    for (int ci = 0; ci < COMPONENT_COUNT; ci++)
    {
        for (int r = 0; r < img->rows; r++)
        {
            int row_index = r * img->cols;
            for (int c = 0; c < img->cols; c++)
            {
                if (compIdxs[row_index + c] == ci)
                {
                    MaskVal m = mask_at(mask, r, c);
                    if (iter == 0)
                    {
                        if (m == GC_BGD || m == GC_PR_BGD)
                        {
                            pixel_t color = {get_r(img, r, c), get_g(img, r, c), get_b(img, r, c)};
                            addSample(bgdGMM, ci, color);
                        }
                        else if (m == GC_FGD || m == GC_PR_FGD)
                        {
                            pixel_t color = {get_r(img, r, c), get_g(img, r, c), get_b(img, r, c)};
                            addSample(fgdGMM, ci, color);
                        }
                    }
                    else
                    {
                        if (m == GC_BGD || m == GC_PR_BGD)
                        {
                            pixel_t color = {get_r(img, r, c), get_g(img, r, c), get_b(img, r, c)};
                            addSample(bgdGMM, ci, color);
                        }
                        else
                        {
                            pixel_t color = {get_r(img, r, c), get_g(img, r, c), get_b(img, r, c)};
                            addSample(fgdGMM, ci, color);
                        }
                    }
                }
            }
        }
    }
    // std::cout << "BGD GMM means weights after learning:" << std::endl;
    endLearning(bgdGMM);
    // std::cout << "FGD GMM means weights after learning:" << std::endl;
    endLearning(fgdGMM);
}

static void constructGCGraph(image_t *img, mask_t *mask, GMM_t *bgdGMM, GMM_t *fgdGMM, double lambda,
                             weight_t leftW, weight_t upleftW, weight_t upW, weight_t uprightW,
                             GCGraph<double> &graph)
{
    if (img == NULL || mask == NULL || bgdGMM == NULL || fgdGMM == NULL)
        return;

    int vtxCount = img->cols * img->rows,
        edgeCount = 2 * (4 * img->cols * img->rows - 3 * (img->cols + img->rows) + 2);

    // cout << "vertex count: " << vtxCount << "\n";
    graph.create(vtxCount, edgeCount);
    // std::cout << "Graph created with " << vtxCount << " vertices and " << edgeCount << " edges." << std::endl;

    // cout << "created graph in construct function\n";
    for (int r = 0; r < img->rows; r++)
    {
        int row_index = r * img->cols;
        for (int c = 0; c < img->cols; c++)
        {
            // add node
            int vtxIdx = graph.addVtx();
            pixel_t color = {get_r(img, r, c), get_g(img, r, c), get_b(img, r, c)};

            // set t-weights
            double fromSource, toSink;
            MaskVal m = mask_at(mask, r, c);
            if (m == GC_PR_BGD || m == GC_PR_FGD)
            {
                fromSource = -log(doSomething(bgdGMM, color) + 1e-6);
                toSink = -log(doSomething(fgdGMM, color) + 1e-6);
            }
            else if (m == GC_BGD)
            {
                fromSource = 0;
                toSink = lambda;
            }
            else // GC_FGD
            {
                fromSource = lambda;
                toSink = 0;
            }
            graph.addTermWeights(vtxIdx, fromSource, toSink);

            // set n-weights
            if (c > 0)
            {
                double w = leftW[row_index + c];
                graph.addEdges(vtxIdx, vtxIdx - 1, w, w);
            }
            if (c > 0 && r > 0)
            {
                double w = upleftW[row_index + c];
                graph.addEdges(vtxIdx, vtxIdx - img->cols - 1, w, w);
            }
            if (r > 0)
            {
                double w = upW[row_index + c];
                graph.addEdges(vtxIdx, vtxIdx - img->cols, w, w);
            }
            if (c < img->cols - 1 && r > 0)
            {
                double w = uprightW[row_index + c];
                graph.addEdges(vtxIdx, vtxIdx - img->cols + 1, w, w);
            }
        }
    }
}

static void estimateSegmentation(GCGraph<double> &graph, mask_t *mask)
{
    int flow = graph.maxFlow();
    // cout << "Max flow: " << flow << "\n";
    for (int r = 0; r < mask->rows; r++)
    {
        for (int c = 0; c < mask->cols; c++)
        {
            MaskVal m = mask_at(mask, r, c);
            if (m == GC_PR_BGD || m == GC_PR_FGD)
            {
                if (graph.inSourceSegment(r * mask->cols + c /*vertex index*/))
                {
                    // cout << "mask[" << r << "][" << c << "] = " << m;
                    mask_set(mask, r, c, GC_PR_FGD);
                    // cout << " mask[" << r << "][" << c << "] = GC_PR_FGD\n";
                }
                else
                {
                    // cout << "mask[" << r << "][" << c << "] = " << m;
                    mask_set(mask, r, c, GC_PR_BGD);
                    // cout << " mask[" << r << "][" << c << "] = GC_PR_BGD\n";
                }
            }
        }
    }
}

void displayImage(image_t *img)
{
    cv::Mat displayImg(img->rows, img->cols, CV_8UC3);
    for (int r = 0; r < img->rows; r++)
    {
        for (int c = 0; c < img->cols; c++)
        {
            displayImg.at<cv::Vec3b>(r, c) = cv::Vec3b(get_b(img, r, c), get_g(img, r, c), get_r(img, r, c));
        }
    }
    cv::imshow("Image", displayImg);
    cv::waitKey(0);
}

void gettingOutput(image_t *img, mask_t *mask, image_t *foreground, image_t *background)
{
    for (int r = 0; r < img->rows; r++)
    {
        for (int c = 0; c < img->cols; c++)
        {
            uint8_t R = get_r(img, r, c);
            uint8_t G = get_g(img, r, c);
            uint8_t B = get_b(img, r, c);
            if (mask_at(mask, r, c) == GC_FGD || mask_at(mask, r, c) == GC_PR_FGD)
                set_rgb(foreground, r, c, R, G, B), set_rgb(background, r, c, 0, 0, 0);
            else
                set_rgb(background, r, c, R, G, B), set_rgb(foreground, r, c, 0, 0, 0);
        }
    }
    // std::cout << "Segmentation result: " << fg << " foreground, " << bg << " background pixels." << std::endl;
}

void grabCut(image_t *img, rect_t rect, image_t *foreground, image_t *background, int iterCount)
{
    int num_pixels = img->rows * img->cols;
    // std::cout << "grabCut called\n";

    GMM_t *bgdGMM, *fgdGMM;
    bgdGMM = (GMM_t *)malloc(sizeof(GMM_t));
    fgdGMM = (GMM_t *)malloc(sizeof(GMM_t));
    mask_t *mask = (mask_t *)malloc(sizeof(mask_t));

    initEmptyGMM(bgdGMM);
    initEmptyGMM(fgdGMM);

    // std::cout << "init GMMs\n";
    int *compIdxs = (int *)malloc(num_pixels * sizeof(int));

    initMaskWithRect(mask, rect, img);
    // gettingOutput(img, mask, foreground, background);
    // displayImage(foreground);
    // displayImage(background);
    //  cout << "After init mask with rect\n";
    initGMMs(img, mask, bgdGMM, fgdGMM);
    // cout << "init gmms again\n";

    if (iterCount <= 0)
        return;

    const double gamma = 50;
    const double lambda = 9 * gamma;

    const double beta = calcBeta(img);

    double *leftW, *upleftW, *upW, *uprightW;
    leftW = (double *)calloc(num_pixels, sizeof(double));
    upleftW = (double *)calloc(num_pixels, sizeof(double));
    upW = (double *)calloc(num_pixels, sizeof(double));
    uprightW = (double *)calloc(num_pixels, sizeof(double));
    calcNWeights(img, leftW, upleftW, upW, uprightW, beta, gamma);

    /*
    std::cout << "Left edge weights sample:" << std::endl;
    for (int y = 0; y < 5; ++y) {
        for (int x = 0; x < 5; ++x) {
            std::cout << leftW[x + (img->cols)*y] << " ";
        }
        std::cout << std::endl;
    } */

    // cout << "After calc nweights\n";
    // std::cout << "Gamma: " << gamma << std::endl;

    // for (int i = 0; i < iterCount; i++) //i< iterCount
    // {
    //     GCGraph<double> graph;
    //     assignGMMsComponents(img, mask, bgdGMM, fgdGMM, compIdxs);
    //     learnGMMs(img, mask, compIdxs, bgdGMM, fgdGMM, i);
    //     constructGCGraph(img, mask, bgdGMM, fgdGMM, lambda, leftW, upleftW, upW, uprightW, graph);
    //     estimateSegmentation(graph, mask);
    // }
    // gettingOutput(img, mask, foreground, background);

    // displayImage(foreground);
    // displayImage(background);
    // cout << "after lop\n";
}

int main()
{
    // st = omp_get_wtime();
    string filepath = "../dataset/small/flower.jpg";
    cv::Mat image = cv::imread(filepath);
    if (image.empty())
    {
        std::cerr << "Image not loaded!" << std::endl;
        return -1;
    }

    std::cout << "Loaded Image " << filepath << std::endl;

    image_t *img = (image_t *)malloc(sizeof(image_t));
    img->rows = image.rows;
    img->cols = image.cols;
    img->r = (uint8_t *)malloc(img->rows * img->cols * sizeof(uint8_t));
    img->g = (uint8_t *)malloc(img->rows * img->cols * sizeof(uint8_t));
    img->b = (uint8_t *)malloc(img->rows * img->cols * sizeof(uint8_t));

    image_t *foreground = (image_t *)malloc(sizeof(image_t));
    image_t *background = (image_t *)malloc(sizeof(image_t));

    foreground->rows = background->rows = image.rows;
    foreground->cols = background->cols = image.cols;

    foreground->r = (uint8_t *)calloc(img->rows * img->cols, sizeof(uint8_t));
    foreground->g = (uint8_t *)calloc(img->rows * img->cols, sizeof(uint8_t));
    foreground->b = (uint8_t *)calloc(img->rows * img->cols, sizeof(uint8_t));
    background->r = (uint8_t *)calloc(img->rows * img->cols, sizeof(uint8_t));
    background->g = (uint8_t *)calloc(img->rows * img->cols, sizeof(uint8_t));
    background->b = (uint8_t *)calloc(img->rows * img->cols, sizeof(uint8_t));

    for (int r = 0; r < img->rows; r++)
    {
        for (int c = 0; c < img->cols; c++)
        {
            cv::Vec3b color = image.at<cv::Vec3b>(r, c);
            set_rgb(img, r, c, color[2], color[1], color[0]);

            /*
            img->array[r * img->cols + c].r = color[2];
            img->array[r * img->cols + c].g = color[1];
            img->array[r * img->cols + c].b = color[0];
            */
        }
    }

    grabCut(img, {132, 75, 845, 525}, foreground, background, 5);
    // 132 75 845 525
    free(img->r);
    free(img->g);
    free(img->b);
    free(img);
    free(foreground->r);
    free(foreground->g);
    free(foreground->b);
    free(foreground);
    free(background->r);
    free(background->g);
    free(background->b);
    free(background);
    return 0;
}